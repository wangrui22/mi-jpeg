#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include "mi_gpu_jpeg_define.h"

#define gpujpeg_div_and_round_up(value, div) \
    ((((value) % (div)) != 0) ? ((value) / (div) + 1) : ((value) / (div)))

inline __device__ void dct_1d_8_fast(
    const float in0, const float in1, const float in2, const float in3, const float in4, const float in5, const float in6, const float in7,
    float & out0, float & out1, float & out2, float & out3, float & out4, float & out5, float & out6, float & out7, const float center_sample = 0.0f) {
    const float diff0 = in0 + in7;
    const float diff1 = in1 + in6;
    const float diff2 = in2 + in5;
    const float diff3 = in3 + in4;
    const float diff4 = in3 - in4;
    const float diff5 = in2 - in5;
    const float diff6 = in1 - in6;
    const float diff7 = in0 - in7;

    const float even0 = diff0 + diff3;
    const float even1 = diff1 + diff2;
    const float even2 = diff1 - diff2;
    const float even3 = diff0 - diff3;

    const float even_diff = even2 + even3;

    const float odd0 = diff4 + diff5;
    const float odd1 = diff5 + diff6;
    const float odd2 = diff6 + diff7;

    const float odd_diff5 = (odd0 - odd2) * 0.382683433f;
    const float odd_diff4 = 1.306562965f * odd2 + odd_diff5;
    const float odd_diff3 = diff7 - odd1 * 0.707106781f;
    const float odd_diff2 = 0.541196100f * odd0 + odd_diff5;
    const float odd_diff1 = diff7 + odd1 * 0.707106781f;

    out0 = even0 + even1 - 8 * center_sample;
    out1 = odd_diff1 + odd_diff4;
    out2 = even3 + even_diff * 0.707106781f;
    out3 = odd_diff3 - odd_diff2;
    out4 = even0 - even1;
    out5 = odd_diff3 + odd_diff2;
    out6 = even3 - even_diff * 0.707106781f;
    out7 = odd_diff1 - odd_diff4;
}

template <int WARP_COUNT>
__global__ void r_2_dct_op_kernel(const BlockUnit rgb, const BlockUnit dct_result, const ImageInfo img_info, const DCTTable dct_table) {
    const int COMPONET = 1;
    __shared__ unsigned char S_YUV[WARP_COUNT*4*64*COMPONET];
    __shared__ float S_QUANT[WARP_COUNT*4*64*COMPONET];
    __shared__ float S_DCT_TABLE[64];
    

    int tid = threadIdx.x;
    int wid = tid/32; //0~3
    int twid = tid - wid*32; //0~31
    int local_mcu_id = twid/8; //0~3
    int mcu_id = blockIdx.x*WARP_COUNT*4 + wid*4 + local_mcu_id;
    int cal_id = twid & 7; //0~7

    S_DCT_TABLE[twid*2] = dct_table.d_quant_tbl_luminance[twid*2];
    S_DCT_TABLE[twid*2+1] = dct_table.d_quant_tbl_luminance[twid*2+1];
    // _S_DCT_TABLE[64+twid*2] = dct_table.d_quant_tbl_chrominance[twid*2];
    // _S_DCT_TABLE[64+twid*2+1] = dct_table.d_quant_tbl_chrominance[twid*2+1];

    // printf("block_id: %d, thread_id: %d, warp_id: %d, wrap_thread_id: %d, local_mcu: %d, mcu: %d, cal_id: %d\n",
    // blockIdx.x, threadIdx.x, wid, twid, local_mcu_id, mcu_id, cal_id);

    if (mcu_id > img_info.mcu_count-1) {
        return; 
    }

    
    int mcu_y = mcu_id / img_info.mcu_w;
    int mcu_x = mcu_id - mcu_y*img_info.mcu_w;
    
    ///\1 rgb->yuv
    int width = img_info.width;
    int height = img_info.height;
    int x0 = mcu_x * 8;
    int y0 = mcu_y * 8;
    int x1 = x0 + 8;
    int y1 = y0 + 8;
    y1 = y1 < height+1 ? y1 : height;
    x1 = x1 < width+1 ? x1 : width;

    unsigned char* s_yuv_base = S_YUV + wid*4*64*COMPONET + local_mcu_id*64*COMPONET + cal_id*COMPONET*8;
    float* s_quant_base = S_QUANT + wid*4*64*COMPONET + local_mcu_id*64*COMPONET;

    int y = y0 + cal_id;
    ((uint*)(s_yuv_base))[0] = 0;
    ((uint*)(s_yuv_base))[1] = 0;
    //补齐0
    // if (y > height-1 ) {
    //     for (int i=0; i<8*COMPONET; ++i) {
    //         s_yuv_base[i] = 0;
    //     }
    // } else {
    //     if (x0 + 8 > height) {
    //         for (int i=0; i<8*COMPONET; ++i) {
    //             s_yuv_base[i] = 0;
    //         }   
    //     }
    //     //赋值
    //     int sidx = 0, idx = 0;
    //     for (int ix=x0; ix<x1; ++ix) {
    //         idx = y*width + ix;
    //         sidx = ix-x0;
    //         s_yuv_base[sidx] = rgb.d_buffer[3*idx];
    //     }
    // } 

    //赋值
    int sidx = 0, idx = 0;
    for (int ix=x0; ix<x1; ++ix) {
        idx = y*width + ix;
        sidx = ix-x0;
        s_yuv_base[sidx] = rgb.d_buffer[3*idx];
    }

    __syncthreads();

    ///\ 2 quantization
    //row 
    float *quant_out0 = s_quant_base + cal_id*COMPONET*8;
    dct_1d_8_fast((float)s_yuv_base[0], (float)s_yuv_base[1], (float)s_yuv_base[2], (float)s_yuv_base[3], 
                  (float)s_yuv_base[4], (float)s_yuv_base[5], (float)s_yuv_base[6], (float)s_yuv_base[7],
                  quant_out0[0], quant_out0[1], quant_out0[2], quant_out0[3], quant_out0[4], quant_out0[5], quant_out0[6], quant_out0[7], 128);
    
    //collumn
    float *quant_out1 = s_quant_base + cal_id;
    dct_1d_8_fast(quant_out1[0], quant_out1[1*8], quant_out1[2*8], quant_out1[3*8], quant_out1[4*8], quant_out1[5*8], quant_out1[6*8], quant_out1[7*8],
                  quant_out1[0], quant_out1[1*8], quant_out1[2*8], quant_out1[3*8], quant_out1[4*8], quant_out1[5*8], quant_out1[6*8], quant_out1[7*8]);

    //write 
    float* tbl = S_DCT_TABLE;
    const int id = cal_id*8;
    int out0 = rintf(quant_out0[0]*tbl[id]);
    int out1 = rintf(quant_out0[1]*tbl[id+1]);
    int out2 = rintf(quant_out0[2]*tbl[id+2]);
    int out3 = rintf(quant_out0[3]*tbl[id+3]);
    int out4 = rintf(quant_out0[4]*tbl[id+4]);
    int out5 = rintf(quant_out0[5]*tbl[id+5]);
    int out6 = rintf(quant_out0[6]*tbl[id+6]);
    int out7 = rintf(quant_out0[7]*tbl[id+7]);
    
    short* quant_write = (short*)dct_result.d_buffer + mcu_id*64*COMPONET + cal_id*COMPONET*8;
    ((uint4*)(quant_write))[0] = make_uint4(
        (out0 & 0xFFFF) + (out1 << 16),
        (out2 & 0xFFFF) + (out3 << 16),
        (out4 & 0xFFFF) + (out5 << 16),
        (out6 & 0xFFFF) + (out7 << 16)
    );
}


extern "C"
hipError_t rgb_2_yuv_2_dct_op(const BlockUnit& rgb, const BlockUnit& dct_result, const ImageInfo& img_info, const DCTTable& dct_table) {
    // const int BLOCK_SIZEX = 4;
    // const int BLOCK_SIZEY = 4;
    // dim3 block(BLOCK_SIZEX, BLOCK_SIZEY, 1);
    // dim3 grid(img_info.mcu_w / BLOCK_SIZEX, img_info.mcu_h / BLOCK_SIZEY);
    // if (grid.x * BLOCK_SIZEX != img_info.mcu_w) {
    //     grid.x += 1;
    // }
    // if (grid.y * BLOCK_SIZEY != img_info.mcu_h) {
    //     grid.y += 1;
    // }

    // kernel_rgb_2_yuv_2_dct << <grid, block >> >(rgb, dct_result, img_info, dct_table);
    
    return hipDeviceSynchronize();
}

extern "C"
hipError_t r_2_dct_op(const BlockUnit& rgb, const BlockUnit& dct_result, const ImageInfo& img_info, const DCTTable& dct_table) {
    const int WARP_COUNT = 4;
    //一个warp32个线程计算4个mcu,一个block计算16个mcu
    dim3 block(32*WARP_COUNT);

    int mcu_count = img_info.mcu_w * img_info.mcu_h;
    dim3 grid = (mcu_count/16);
    if (grid.x * 16 != mcu_count) {
        grid.x += 1;
    }

    r_2_dct_op_kernel<WARP_COUNT> <<<grid, block>>>(rgb, dct_result, img_info, dct_table);
    
    return hipDeviceSynchronize();
}
