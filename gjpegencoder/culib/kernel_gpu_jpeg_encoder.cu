#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include "mi_gpu_jpeg_define.h"

template<typename T0, typename T>
inline __device__ void dct_1d_8_fast(const T0 in0, const T0 in1, const T0 in2, const T0 in3, const T0 in4, const T0 in5, const T0 in6, const T0 in7,
    T & out0, T & out1, T & out2, T & out3, T & out4, T & out5, T & out6, T & out7, const float center_sample = 0.0f) {
    const float diff0 = in0 + in7;
    const float diff1 = in1 + in6;
    const float diff2 = in2 + in5;
    const float diff3 = in3 + in4;
    const float diff4 = in3 - in4;
    const float diff5 = in2 - in5;
    const float diff6 = in1 - in6;
    const float diff7 = in0 - in7;

    const float even0 = diff0 + diff3;
    const float even1 = diff1 + diff2;
    const float even2 = diff1 - diff2;
    const float even3 = diff0 - diff3;

    const float even_diff = even2 + even3;

    const float odd0 = diff4 + diff5;
    const float odd1 = diff5 + diff6;
    const float odd2 = diff6 + diff7;

    const float odd_diff5 = (odd0 - odd2) * 0.382683433f;
    const float odd_diff4 = 1.306562965f * odd2 + odd_diff5;
    const float odd_diff3 = diff7 - odd1 * 0.707106781f;
    const float odd_diff2 = 0.541196100f * odd0 + odd_diff5;
    const float odd_diff1 = diff7 + odd1 * 0.707106781f;

    out0 = even0 + even1 - 8 * center_sample;
    out1 = odd_diff1 + odd_diff4;
    out2 = even3 + even_diff * 0.707106781f;
    out3 = odd_diff3 - odd_diff2;
    out4 = even0 - even1;
    out5 = odd_diff3 + odd_diff2;
    out6 = even3 - even_diff * 0.707106781f;
    out7 = odd_diff1 - odd_diff4;
}


__shared__ unsigned char _S_ZIG_ZAG[64];
__shared__ float _S_DCT_TABLE[128];

inline __device__ uint8_t gpujpeg_clamp(int value) {
    value = (value >= 0) ? value : 0;
    value = (value <= 255) ? value : 255;
    return (uint8_t)value;
}

template<int bit_depth> inline __device__ void
gpujpeg_color_transform_to(uint8_t & c1, uint8_t & c2, uint8_t & c3, const int matrix[9], int base1, int base2, int base3)
{
    // Prepare integer constants
    const int middle = 1 << (bit_depth - 1);

    // Perform color transform
    int r1 = (int)c1 * 256 / 255;
    int r2 = (int)c2 * 256 / 255;
    int r3 = (int)c3 * 256 / 255;
    c1 = gpujpeg_clamp(((matrix[0] * r1 + matrix[1] * r2 + matrix[2] * r3 + middle) >> bit_depth) + base1);
    c2 = gpujpeg_clamp(((matrix[3] * r1 + matrix[4] * r2 + matrix[5] * r3 + middle) >> bit_depth) + base2);
    c3 = gpujpeg_clamp(((matrix[6] * r1 + matrix[7] * r2 + matrix[8] * r3 + middle) >> bit_depth) + base3);
}

__device__ void rgb_2_yuv_unit(uint8_t & c1, uint8_t & c2, uint8_t & c3) {
    /*const double matrix[] = {
          0.299000,  0.587000,  0.114000,
         -0.147400, -0.289500,  0.436900,
          0.615000, -0.515000, -0.100000
    };*/
    const int matrix[] = {77, 150, 29, -38, -74, 112, 157, -132, -26};
    gpujpeg_color_transform_to<8>(c1, c2, c3, matrix, 0, 128, 128);
}


__global__ void kernel_rgb_2_yuv_2_dct(const BlockUnit rgb, const BlockUnit dct_result, const ImageInfo img_info, const DCTTable dct_table) {
    unsigned int mcu_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int mcu_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (mcu_x > img_info.mcu_w-1 || mcu_y > img_info.mcu_h-1) {
        return;
    }
    if (threadIdx.x == 0) {
        for (int i=0; i<64; ++i) {
            _S_ZIG_ZAG[i] = dct_table.d_zig_zag[i];
            _S_DCT_TABLE[i] = dct_table.d_quant_tbl_luminance[i];
            _S_DCT_TABLE[i+64] = dct_table.d_quant_tbl_chrominance[i];
        }
    }
    __syncthreads();

    int mcu_id = mcu_y*img_info.mcu_w + mcu_x;

    int width = img_info.width;
    int height = img_info.height;
    //int width_ext = img_info.width_ext;
    //int height_ext = img_info.height_ext;

    int x0 = mcu_x * 8;
    int y0 = mcu_y * 8;
    int x1 = x0 + 8;
    int y1 = y0 + 8;
    y1 = y1 < height ? y1 : height;
    x1 = x1 < width ? x1 : width;

    int sidx = 0;
    int idx = 0;
    float r,g,b,y,u,v;
    //unsigned char r,g,b;
    unsigned char yuv[64*3];
    for (int i=0; i<64*3; ++i) {
        yuv[i] = 0;
    }
    for (int iy=y0; iy<y1; ++iy) {
        for (int ix=x0; ix<x1; ++ix) {
            idx = iy*width + ix;
            sidx = (iy-y0)*8 + (ix-x0);
            r = (float)rgb.d_buffer[idx*3];
            g = (float)rgb.d_buffer[idx*3+1];
            b = (float)rgb.d_buffer[idx*3+2];
            y =  0.2990f*r + 0.5870f*g + 0.1140f*b ;
            u = -0.1687f*r - 0.3313f*g + 0.5000f*b + 128.0f;
            v =  0.5000f*r - 0.4187f*g - 0.0813f*b + 128.0f;
            y = y < 0.0f ? 0.0f : y;
            y = y > 255.0f ? 255.0f : y;
            u = u < 0.0f ? 0.0f : u;
            u = u > 255.0f ? 255.0f : u;
            v = v < 0.0f ? 0.0f : v;
            v = v > 255.0f ? 255.0f : v;

            yuv[sidx] = (unsigned char)y;
            yuv[sidx+64] = (unsigned char)u;
            yuv[sidx+128] = (unsigned char)v;
            
            //和调用rgb_2_yuv_unit性能上差别不大
            // r = rgb.d_buffer[idx*3];
            // g = rgb.d_buffer[idx*3+1];
            // b = rgb.d_buffer[idx*3+2];
            // rgb_2_yuv_unit(r,g,b);
            // yuv[sidx] = r;
            // yuv[sidx+64] = g;
            // yuv[sidx+128] = b;

        }
    }

    float quant_local[64];
    short *quant_base = (short*)dct_result.d_buffer + mcu_id*64*3;
    // float *tbls[3] = {dct_table.d_quant_tbl_luminance, dct_table.d_quant_tbl_chrominance, dct_table.d_quant_tbl_chrominance};
    // unsigned char* ZIGZAG_TABLE = dct_table.d_zig_zag;
    float *tbls[3] = {_S_DCT_TABLE, _S_DCT_TABLE+64, _S_DCT_TABLE+64};
    unsigned char* ZIGZAG_TABLE = _S_ZIG_ZAG;

    for (int j=0; j<3; ++j) {
        short *quant_val = quant_base + 64*j;
        unsigned char *val = yuv + 64*j;
        float* tbl = tbls[j];

        for (int i=0; i<8; ++i) {
            unsigned char* i0 = val + 8*i;
            float* o0 = quant_local + 8*i;
            dct_1d_8_fast<unsigned char, float>(i0[0], i0[1], i0[2], i0[3], i0[4], i0[5], i0[6], i0[7],
                        o0[0], o0[1], o0[2], o0[3], o0[4], o0[5], o0[6], o0[7], 128);
        }

        for (int i=0; i<8; ++i) {
            float* i0 = quant_local + i;
            float* o0 = quant_local + i;
            dct_1d_8_fast<float, float>(i0[0], i0[1*8], i0[2*8], i0[3*8], i0[4*8], i0[5*8], i0[6*8], i0[7*8],
                        o0[0], o0[1*8], o0[2*8], o0[3*8], o0[4*8], o0[5*8], o0[6*8], o0[7*8], 0);
        }

        for (int i=0; i<64; ++i) {
            float v = quant_local[i]*tbl[i];
            if (v < 0.0f) {
                v-=0.5f;
            } else {
                v+=0.5f;
            }
            quant_val[ZIGZAG_TABLE[i]] = (short)v;
        }
    }

}

__global__ void kernel_r_2_dct(const BlockUnit rgb, const BlockUnit dct_result, const ImageInfo img_info, const DCTTable dct_table) {
    unsigned int mcu_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int mcu_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (mcu_x > img_info.mcu_w-1 || mcu_y > img_info.mcu_h-1) {
        return;
    }
    const int component = img_info.component;
    if (threadIdx.x == 0) {
        for (int i=0; i<64; ++i) {
            _S_ZIG_ZAG[i] = dct_table.d_zig_zag[i];
            _S_DCT_TABLE[i] = dct_table.d_quant_tbl_luminance[i];
            //_S_DCT_TABLE[i+64] = dct_table.d_quant_tbl_chrominance[i];
        }
    }
    __syncthreads();

    int mcu_id = mcu_y*img_info.mcu_w + mcu_x;

    int width = img_info.width;
    int height = img_info.height;
    //int width_ext = img_info.width_ext;
    //int height_ext = img_info.height_ext;

    int x0 = mcu_x * 8;
    int y0 = mcu_y * 8;
    int x1 = x0 + 8;
    int y1 = y0 + 8;
    y1 = y1 < height ? y1 : height;
    x1 = x1 < width ? x1 : width;

    int sidx = 0;
    int idx = 0;
    unsigned char yuv[64];
    for (int i=0; i<64; ++i) {
        yuv[i] = 0;
    }
    for (int iy=y0; iy<y1; ++iy) {
        for (int ix=x0; ix<x1; ++ix) {
            idx = iy*width + ix;
            sidx = (iy-y0)*8 + (ix-x0);
            yuv[sidx] =rgb.d_buffer[idx*3];
        }
    }

    float quant_local[64];
    short *quant_base = (short*)dct_result.d_buffer + mcu_id*64*component;
    unsigned char* ZIGZAG_TABLE = _S_ZIG_ZAG;

    //for (int j=0; j<3; ++j) {
        short *quant_val = quant_base;
        unsigned char *val = yuv;
        float* tbl = _S_DCT_TABLE;

        for (int i=0; i<8; ++i) {
            unsigned char* i0 = val + 8*i;
            float* o0 = quant_local + 8*i;
            dct_1d_8_fast<unsigned char, float>(i0[0], i0[1], i0[2], i0[3], i0[4], i0[5], i0[6], i0[7],
                        o0[0], o0[1], o0[2], o0[3], o0[4], o0[5], o0[6], o0[7], 128);
        }

        for (int i=0; i<8; ++i) {
            float* i0 = quant_local + i;
            float* o0 = quant_local + i;
            dct_1d_8_fast<float, float>(i0[0], i0[1*8], i0[2*8], i0[3*8], i0[4*8], i0[5*8], i0[6*8], i0[7*8],
                        o0[0], o0[1*8], o0[2*8], o0[3*8], o0[4*8], o0[5*8], o0[6*8], o0[7*8], 0);
        }

        for (int i=0; i<64; ++i) {
            // float v = quant_local[i]*tbl[i];
            // if (v < 0.0f) {
            //     v-=0.5f;
            // } else {
            //     v+=0.5f;
            // }
            // quant_val[ZIGZAG_TABLE[i]] = (short)v;

            quant_val[ZIGZAG_TABLE[i]] = (short)rintf(quant_local[i]*tbl[i]);
        }
    //}

}


__device__ BitString get_bit_code(int value) {
	int v = (value > 0) ? value : -value;
	int length = 0;
	for(length = 0; v>0; v >>= 1) {
        length++;
    }

    BitString ret;
	ret.value = value > 0 ? value : (1 << length) + value - 1;
	ret.length = length;
	return ret;
}


__shared__ BitString _S_huffman_table_Y_DC[12];
__shared__ BitString _S_huffman_table_Y_AC[256];
__shared__ BitString _S_huffman_table_CbCr_DC[12];
__shared__ BitString _S_huffman_table_CbCr_AC[256];

__global__ void kernel_huffman_encoding(const BlockUnit dct_result, const BlockUnit huffman_code, int *d_huffman_code_count, const ImageInfo img_info, const HuffmanTable huffman_table) {
    unsigned int mcu_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int mcu_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (mcu_x > img_info.mcu_w-1 || mcu_y > img_info.mcu_h-1) {
        return;
    }
    const int component = img_info.component;
    // if (threadIdx.x == 0) {
    //     for (int i=0; i<12; ++i) {
    //         _S_huffman_table_Y_DC[i] = huffman_table.d_huffman_table_Y_DC[i];
    //         _S_huffman_table_CbCr_DC[i] = huffman_table.d_huffman_table_CbCr_DC[i];
    //     }
    //     for (int i=0; i<256; ++i) {
    //         _S_huffman_table_Y_AC[i] = huffman_table.d_huffman_table_Y_AC[i];
    //         _S_huffman_table_CbCr_AC[i] = huffman_table.d_huffman_table_CbCr_AC[i];
    //     }
        
    // }
    // __syncthreads();

    int mcu_id = mcu_y*img_info.mcu_w + mcu_x;

    int width = img_info.width;
    int height = img_info.height;
    //int width_ext = img_info.width_ext;
    //int height_ext = img_info.height_ext;

    int x0 = mcu_x * 8;
    int y0 = mcu_y * 8;
    int x1 = x0 + 8;
    int y1 = y0 + 8;
    y1 = y1 < height ? y1 : height;
    x1 = x1 < width ? x1 : width;


    BitString* HTDCs[3] = {huffman_table.d_huffman_table_Y_DC, huffman_table.d_huffman_table_CbCr_DC, huffman_table.d_huffman_table_CbCr_DC};
    BitString* HTACs[3] = {huffman_table.d_huffman_table_Y_AC, huffman_table.d_huffman_table_CbCr_AC, huffman_table.d_huffman_table_CbCr_AC};
    // BitString* HTDCs[3] = {_S_huffman_table_Y_DC, _S_huffman_table_CbCr_DC, _S_huffman_table_CbCr_DC};
    // BitString* HTACs[3] = {_S_huffman_table_Y_AC, _S_huffman_table_CbCr_AC, _S_huffman_table_CbCr_AC};

    short *quant_base = (short*)dct_result.d_buffer + mcu_id*64*component;
    BitString* output_base = (BitString*)huffman_code.d_buffer + mcu_id*256*component;
    int* output_count = d_huffman_code_count + mcu_id*component;
    
    int segment_id = mcu_id/img_info.segment_mcu_count;
    int mcu_id_in_seg = mcu_id - segment_id*img_info.segment_mcu_count;
    short preDC[3] = {0,0,0};
    if (component == 3) {
        if (mcu_id_in_seg != 0) {
            preDC[0] = *(quant_base-64*3);
            preDC[1] = *(quant_base-64*2);
            preDC[2] = *(quant_base-64);
        }
    } else {
        if (mcu_id_in_seg != 0) {
            preDC[0] = *(quant_base-64);
        }
    }
    

    for (int j=0; j<component; ++j) {
        short *quant = quant_base + 64*j;
        BitString *output = output_base + 256*j;
        BitString *HTDC = HTDCs[j];
        BitString *HTAC = HTACs[j];

        int index = 0;
        //encode DC
        const int diffDC = quant[0] - preDC[j];
        
        if (0 == diffDC) {
            output[index++] = HTDC[0];
        } else {
            BitString bs = get_bit_code(diffDC);
            output[index++] = HTDC[bs.length];
            output[index++] = bs;
        }

        //encode AC
        BitString EOB = HTAC[0x00];
        BitString SIXTEEN_ZEROS = HTAC[0xF0];

        int end_pos = 63;
        while (end_pos > 0 && quant[end_pos] == 0 ) {
            --end_pos;
        }

        for (int i=1; i<=end_pos; ) {
            int start_pos = i;
            while(quant[i] == 0 && i <= end_pos) {
                ++i;
            }

            int zero_counts = i - start_pos;
            if (zero_counts >= 16) {
                for (int j=0; j < zero_counts/16; ++j)
                    output[index++] = SIXTEEN_ZEROS;
                zero_counts = zero_counts%16;
            }

            BitString bs = get_bit_code(quant[i]);

            output[index++] = HTAC[(zero_counts << 4) | bs.length];
            output[index++] = bs;
            i++;
        }

        if (end_pos != 63) {
            output[index++] = EOB;
        }

        output_count[j] = index;

        if (index >= 256) {
            printf("err\n");
        }
    }

}

__device__ void write_byte(unsigned char val, unsigned char* buffer, int& byte) {
    *buffer = val;
    byte += 1;
}

__device__ void write_bitstring(const BitString* bs, int counts, int& new_byte, int& new_byte_pos, unsigned char* buffer, int& byte) {
    const unsigned short mask[] = {1,2,4,8,16,32,64,128,256,512,1024,2048,4096,8192,16384,32768};
	for(int i=0; i<counts; ++i) {
		int value = bs[i].value;
		int posval = bs[i].length - 1;
		while (posval >= 0) {
			if ((value & mask[posval]) != 0) {
				new_byte = new_byte  | mask[new_byte_pos];
			}
			posval--;
			new_byte_pos--;
			if (new_byte_pos < 0) {
				write_byte((unsigned char)(new_byte), buffer++, byte);
				if (new_byte == 0xFF){
					//special case
					write_byte((unsigned char)(0x00), buffer++, byte);
				}
				new_byte_pos = 7;
				new_byte = 0;
			}
		}
	}
}

__global__ void kernel_huffman_writebits(const BlockUnit huffman_code, int *d_huffman_code_count, const ImageInfo img_info, BlockUnit segment_compressed, int *d_segment_compressed_byte) {
    unsigned int segid = blockIdx.x * blockDim.x + threadIdx.x;
    if (segid > img_info.segment_count-1) {
        return;
    }
    const int component = img_info.component;

    const unsigned int MCU_HUFFMAN_CAPACITY = 256;
    const int MAX_SEGMENT_BYTE = 4096;

    const int mcu0 = segid*img_info.segment_mcu_count;
    int mcu1 = mcu0 + img_info.segment_mcu_count;
    if (mcu1 > img_info.mcu_count-1) {
        mcu1 = img_info.mcu_count-1;
    }

    unsigned char* buffer = segment_compressed.d_buffer + segid*MAX_SEGMENT_BYTE;
    int segment_compressed_byte = 0;
    
    int new_byte=0, new_byte_pos=7;
    for (int m=mcu0; m<mcu1; ++m) {
        BitString* huffman_code_seg = (BitString*)huffman_code.d_buffer+MCU_HUFFMAN_CAPACITY*component*m;
        int* huffman_code_count_seg = d_huffman_code_count+component*m;
        for (int i=0; i<component; ++i) {
            write_bitstring(huffman_code_seg, *huffman_code_count_seg, new_byte, new_byte_pos, buffer+segment_compressed_byte, segment_compressed_byte);
            huffman_code_seg += MCU_HUFFMAN_CAPACITY;
            huffman_code_count_seg += 1;    
        }
        // write_bitstring(huffman_code_seg, *huffman_code_count_seg, new_byte, new_byte_pos, buffer+segment_compressed_byte, segment_compressed_byte);
        // huffman_code_seg += MCU_HUFFMAN_CAPACITY;
        // huffman_code_count_seg += 1;
        // write_bitstring(huffman_code_seg, *huffman_code_count_seg, new_byte, new_byte_pos, buffer+segment_compressed_byte, segment_compressed_byte);
        // huffman_code_seg += MCU_HUFFMAN_CAPACITY;
        // huffman_code_count_seg += 1;
        // write_bitstring(huffman_code_seg, *huffman_code_count_seg, new_byte, new_byte_pos, buffer+segment_compressed_byte, segment_compressed_byte);
    }
    if (new_byte_pos != 7) {
        int bp = new_byte_pos;
        int b = new_byte; 
        int mask[8] = {1,2,4,8,16,32,64,128};
        while (bp>=0) {
            b = b | mask[bp];                
            --bp;
        }
        write_byte((unsigned char)b, buffer+segment_compressed_byte, segment_compressed_byte);
        new_byte_pos = 7;
        new_byte = 0;
    }

    write_byte(0xFF, buffer+segment_compressed_byte, segment_compressed_byte);
    write_byte(0xD0+segid%8, buffer+segment_compressed_byte, segment_compressed_byte);

    if (segment_compressed_byte > 4095) {
        printf("segment byte error: %d\n", segment_compressed_byte);   
    }
    d_segment_compressed_byte[segid] = segment_compressed_byte;
}

__global__ void kernel_segment_offset(const ImageInfo img_info, int *d_segment_compressed_byte, int *d_segment_compressed_offset)  {
    unsigned int segid = blockIdx.x * blockDim.x + threadIdx.x;
    if (segid > img_info.segment_count-1) {
        return;
    }
    int val = 0;
    for (int i=0; i<segid; ++i) {
        if (d_segment_compressed_byte[i] < 0) {
            //printf("segment byte error: %d\n", d_segment_compressed_byte[i]);    
        }
        val += d_segment_compressed_byte[i];
    }
    d_segment_compressed_offset[segid] = val;
    if (segid == img_info.segment_count-1) {
        printf("last segment offset: %d\n", val);
    }
}

__global__ void kernel_segment_compact(const BlockUnit segment_compressed, const ImageInfo img_info, int *d_segment_compressed_byte, int *d_segment_compressed_offset, const BlockUnit segment_compressed_compact) {
    unsigned int segid = blockIdx.x * blockDim.x + threadIdx.x;
    if (segid > img_info.segment_count-1) {
        return;
    }  
    const int MAX_SEGMENT_BYTE = 4096;
    const int src_offset = segid*MAX_SEGMENT_BYTE;
    const int dst_offset = d_segment_compressed_offset[segid];
    const int len = d_segment_compressed_byte[segid];
    unsigned char* src = segment_compressed.d_buffer + src_offset;
    unsigned char* dst = segment_compressed_compact.d_buffer + dst_offset;
    for (int i=0; i<len; ++i) {
        dst[i] = src[i];
    }
}

extern "C"
hipError_t rgb_2_yuv_2_dct(const BlockUnit& rgb, const BlockUnit& dct_result, const ImageInfo& img_info, const DCTTable& dct_table) {
    const int BLOCK_SIZEX = 4;
    const int BLOCK_SIZEY = 4;
    dim3 block(BLOCK_SIZEX, BLOCK_SIZEY, 1);
    dim3 grid(img_info.mcu_w / BLOCK_SIZEX, img_info.mcu_h / BLOCK_SIZEY);
    if (grid.x * BLOCK_SIZEX != img_info.mcu_w) {
        grid.x += 1;
    }
    if (grid.y * BLOCK_SIZEY != img_info.mcu_h) {
        grid.y += 1;
    }

    kernel_rgb_2_yuv_2_dct << <grid, block >> >(rgb, dct_result, img_info, dct_table);
    
    return hipDeviceSynchronize();
}

extern "C"
hipError_t r_2_dct(const BlockUnit& rgb, const BlockUnit& dct_result, const ImageInfo& img_info, const DCTTable& dct_table) {
    const int BLOCK_SIZEX = 4;
    const int BLOCK_SIZEY = 4;
    dim3 block(BLOCK_SIZEX, BLOCK_SIZEY, 1);
    dim3 grid(img_info.mcu_w / BLOCK_SIZEX, img_info.mcu_h / BLOCK_SIZEY);
    if (grid.x * BLOCK_SIZEX != img_info.mcu_w) {
        grid.x += 1;
    }
    if (grid.y * BLOCK_SIZEY != img_info.mcu_h) {
        grid.y += 1;
    }

    kernel_r_2_dct << <grid, block >> >(rgb, dct_result, img_info, dct_table);
    
    return hipDeviceSynchronize();
}

extern "C"
hipError_t huffman_encoding(const BlockUnit& dct_result, const BlockUnit& huffman_code, int *d_huffman_code_count, const ImageInfo& img_info, const HuffmanTable& huffman_table) {
    const int BLOCK_SIZEX = 4;
    const int BLOCK_SIZEY = 4;
    dim3 block(BLOCK_SIZEX, BLOCK_SIZEY, 1);
    dim3 grid(img_info.mcu_w / BLOCK_SIZEX, img_info.mcu_h / BLOCK_SIZEY);
    if (grid.x * BLOCK_SIZEX != img_info.mcu_w) {
        grid.x += 1;
    }
    if (grid.y * BLOCK_SIZEY != img_info.mcu_h) {
        grid.y += 1;
    }

    kernel_huffman_encoding << <grid, block >> >(dct_result, huffman_code, d_huffman_code_count, img_info, huffman_table);
    
    return hipDeviceSynchronize();
}

extern "C"
hipError_t huffman_writebits(const BlockUnit& huffman_code, int *d_huffman_code_count, const ImageInfo& img_info, const BlockUnit& segment_compressed, int *d_segment_compressed_byte) {
    const int BLOCK_SIZE = 8;
    dim3 block(BLOCK_SIZE, 1, 1);
    dim3 grid(img_info.segment_count / BLOCK_SIZE, 1, 1);
    if (grid.x * BLOCK_SIZE != img_info.segment_count) {
        grid.x += 1;
    }

    kernel_huffman_writebits << <grid, block >> >(huffman_code, d_huffman_code_count, img_info, segment_compressed, d_segment_compressed_byte);
    
    return hipDeviceSynchronize();
}

extern "C"
hipError_t segment_offset(const ImageInfo& img_info, int *d_segment_compressed_byte, int *d_segment_compressed_offset) {
    const int BLOCK_SIZE = 8;
    dim3 block(BLOCK_SIZE, 1, 1);
    dim3 grid(img_info.segment_count / BLOCK_SIZE, 1, 1);
    if (grid.x * BLOCK_SIZE != img_info.segment_count) {
        grid.x += 1;
    }

    kernel_segment_offset << <grid, block >> >(img_info, d_segment_compressed_byte, d_segment_compressed_offset);
    
    return hipDeviceSynchronize();
}

extern "C"
hipError_t segment_compact(const BlockUnit& segment_compressed, const ImageInfo& img_info, int *d_segment_compressed_byte, int *d_segment_compressed_offset, const BlockUnit& segment_compressed_compact) {
    const int BLOCK_SIZE = 8;
    dim3 block(BLOCK_SIZE, 1, 1);
    dim3 grid(img_info.segment_count / BLOCK_SIZE, 1, 1);
    if (grid.x * BLOCK_SIZE != img_info.segment_count) {
        grid.x += 1;
    }

    kernel_segment_compact << <grid, block >> >(segment_compressed, img_info, d_segment_compressed_byte, d_segment_compressed_offset, segment_compressed_compact);
    
    return hipDeviceSynchronize();
}